#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "pooling.h"
#include "hip/hip_runtime.h"
}

void forward_pool_gpu(pool_layer pl, float *input_gpu) {
  float alpha = 1.0f;
  float beta = 0.0f;
  CUDNN_CHECK(hipdnnPoolingForward(cudnn_handler(),
                          pl.poolDesc,
                          &alpha,
                          pl.inputTensorDesc,
                          input_gpu,
                          &beta,
                          pl.outputTensorDesc,
                          pl.output_gpu));
}
