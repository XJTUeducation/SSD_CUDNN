#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "pooling.h"
#include "hip/hip_runtime.h"
}

void forward_pool_layer_gpu(pool_layer pl, float *input_gpu) {
  float *alpha = 1.0;
  float *beta = 0.0;
  CUDNN_CHECK(hipdnnPoolingForward(cudnn_handler(),
                          pl.poolDesc,
                          &alpha,
                          pl.inputTensorDesc,
                          &beta,
                          pl.outputTensorDesc,
                          pl.output_gpu));
}
