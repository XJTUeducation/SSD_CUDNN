#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "l2_normalize.h"
#include "hip/hip_runtime.h"
}

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>


__global__ void addKernel(float *input) {
  const int id = blockDim.x * blockIdx.x + threadIdx.x;
  
}

void forward_normalize_gpu(normalize_layer nl, float *input_gpu) {
  float alpha = 1.0f;
  float beta = 0.0f;
  // Sum powed matrix to input channel per Batch
  // Pow 2 for whole batch.

  // Caluculate Sum for input/output channel and save. Shape is HW.
  CUBLAS_CHECK(hipblasSgemv(cublas_handler(), HIPBLAS_OP_N, nl.out_c,
               nl.out_h * nl.out_w, &alpha, input_gpu, nl.out_c, nl.ones_channel_gpu,
               1, &beta, nl.out_norm_gpu, 1));

  // Pow 1/2 for each batch.

  // TODO 統合できるかも？
  // divide each batch(CHW) by calculated normalize vector(HW).

  // Scale to whole batch.

}
