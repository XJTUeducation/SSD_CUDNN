#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "element.h"
#include "hip/hip_runtime.h"
}

void element_max_gpu(int *output, float *input, int size, int incx) {
  CUBLAS_CHECK(hipblasIsamax(cublas_handler(), size, input, incx, output));
  // transpose_kernel<<<block, grid>>>(output, input, w, h);
}
